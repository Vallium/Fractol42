
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
# define WIN_SZ_X 1024
# define WIN_SZ_Y 1024

__global__ void			mandelbrot(int	*d_i, double offx, double offy, double zoom, int ite_max, int winszx, int winszy)
{
	double	x1;
	double	y1;
	double	c_r;
	double	c_i;
	double	z_r;
	double	z_i;
	double	tmp;
	int		i;
	int		row;  // WIDTH
	int		col;  // HEIGHT
	int		index;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	col = blockIdx.x * blockDim.x + threadIdx.x;
	index = row * winszx + col;
	if(col >= winszx || row >= winszy)
		return;

	x1 = -2.1;
	y1 = -1.2;
	c_r = (((double)col + offx) / zoom) + x1;
	c_i = (((double)row + offy) / zoom) + y1;
	z_r = 0.0;
	z_i = 0.0;
	i = 0;
	while((z_r * z_r + z_i * z_i) < 4 && i < ite_max)
	{
		tmp = z_r;
		z_r = (z_r * z_r) - (z_i * z_i) + c_r;
		z_i = (2 * tmp * z_i) + c_i;
		i++;
	}
	d_i[index] = i;
}

extern "C" void			call_mandelbrot(int *i, double offx, double offy, double zoom, int ite_max, int winszx, int winszy)
{
	int		*d_i;
	int		size;
	dim3	block_size(16, 16);
	dim3	grid_size(WIN_SZ_X / block_size.x, WIN_SZ_Y / block_size.y);

	size = WIN_SZ_Y * WIN_SZ_X * sizeof(int);
	hipMalloc((void **)&d_i, size);
	mandelbrot<<<grid_size,block_size>>>(d_i, offx, offy, zoom, ite_max, winszx, winszy);

	hipMemcpy(i, d_i, size, hipMemcpyDeviceToHost);
	hipFree(d_i);
}
