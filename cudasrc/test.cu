#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (2048 * 2048)
#define M 512

__global__ void			madelbrot(int	*d_i, t_pos *pt, t_all *all)
{
	double	x1;
	double	y1;
	double	c_r;
	double	c_i;
	double	z_r;
	double	z_i;
	double	tmp;
	int		i;
	int		row;  // WIDTH
	int		col;  // HEIGHT
	int		idx;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	col = blockIdx.x * blockDim.x + threadIdx.x;
	index = row * WIN_SZ_X + col;
	if(col >= WIN_SZ_X || row >= WIN_SZ_Y)
		return;

	x1 = -2.1;
	y1 = -1.2;
	c_r = (((double)pt->x + (double)all->off.x) / (double)all->zoom) + x1;
	c_i = (((double)pt->y + (double)all->off.y) / (double)all->zoom) + y1;
	z_r = 0.0;
	z_i = 0.0;
	i = 0;
	while((z_r * z_r + z_i * z_i) < 4 && i < all->ite_max)
	{
		tmp = z_r;
		z_r = (z_r * z_r) - (z_i * z_i) + c_r;
		z_i = (2 * tmp * z_i) + c_i;
		i++;
	}
	d_i[index] = i;
}

extern "C" void			call_mandelbrot(t_all *all)
{
	t_pos	*pt;
	int		i;
	int		*d_i;
	dim3	block_size(16, 16);
	dim3	grid_size(WIN_SZ_X / block_size.x, WIN_SZ_Y / block_size.y);

	size = WIN_SZ_Y * WIN_SZ_X * sizeof(int);
	i = 0;
	hipMalloc((void **)&d_i, size);
	pt = (t_pos *)malloc(sizeof(t_pos));
	pt->x = 0;
	while (pt->x < WIN_SZ_X)
	{
		pt->y = 0;
		while (pt->y < WIN_SZ_X)
		{
			mandelbrot<<<grid_size,block_size>>>(d_i, pt, all);
			hipMemcpy(&i, d_i, size, hipMemcpyDeviceToHost);
			rainbow_color((double)i / (double)all->ite_max, all);
			ft_put_pxl(all, pt);
		}
	}
}
