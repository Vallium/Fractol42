
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include "cuda_call.h"
#include <stdio.h>
#define N 512

__global__ void		add(double *a, double *b, double *c)
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

extern "C" void		call_add(void)
{
	double *a, *b, *c;
	double *d_a, *d_b, *d_c;
	double size = N * sizeof(double);
	int		i;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (double *)malloc(size);
	b = (double *)malloc(size);
	c = (double *)malloc(size);

	for (i = 0; i < N; ++i)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<1,N>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (i = 0; i < N; ++i)
		printf("%f + %f = %f\n", a[i], b[i], c[i]);
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
//	return (0);
}
