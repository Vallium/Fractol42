
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (2048 * 2048)
#define M 512

__global__ void		add(double *a, double *b, double *c, int n)
{
	int		index;

	index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
		c[index] = a[index] + b[index];
}

extern "C" void		call_add(char * str)
{
	double *a, *b, *c;
	double *d_a, *d_b, *d_c;
	double size = N * sizeof(double);
	int		i;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (double *)malloc(size);
	b = (double *)malloc(size);
	c = (double *)malloc(size);

	for (i = 0; i < N; ++i)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<(N + M - 1) / M,M>>>(d_a, d_b, d_c, N);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (i = 0; i < N; ++i)
		printf("%s -> %f + %f = %f\n", str, a[i], b[i], c[i]);
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
