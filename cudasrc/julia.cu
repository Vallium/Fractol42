
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
# define WIN_SZ_X 1024
# define WIN_SZ_Y 1024

__global__ void		julia(int	*d_i, double offx, double offy, double zoom, int ite_max, int winszx, int winszy, double c_i, double c_r)
{
	double	x1;
	double	y1;
	double	z_r;
	double	z_i;
	double	tmp;
	int		i;
	int		row;  // WIDTH
	int		col;  // HEIGHT
	int		index;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	col = blockIdx.x * blockDim.x + threadIdx.x;
	index = row * winszx + col;
	if(col >= winszx || row >= winszy)
		return;

	x1 = -1.4;
	y1 = -1.2;
	z_r = (((double)col + (double)offx) / (double)zoom) + x1;
	z_i = (((double)row + (double)offy) / (double)zoom) + y1;
	i = 0;
	while((z_r * z_r + z_i * z_i) < 4 && i < ite_max)
	{
		tmp = z_r;
		z_r = (z_r * z_r) - (z_i * z_i) + c_r;
		z_i = (2 * tmp * z_i) + c_i;
		i++;
	}
	d_i[index] = i;
}

extern "C" void		call_julia(int *i, double offx, double offy, double zoom, int ite_max, int winszx, int winszy, double c_i, double c_r)
{
	int		*d_i;
	int		size;
	dim3	block_size(16, 16);
	dim3	grid_size(WIN_SZ_X / block_size.x, WIN_SZ_Y / block_size.y);

	size = WIN_SZ_Y * WIN_SZ_X * sizeof(int);
	hipMalloc((void **)&d_i, size);
	julia<<<grid_size,block_size>>>(d_i, offx, offy, zoom, ite_max, winszx, winszy, c_i, c_r);

	hipMemcpy(i, d_i, size, hipMemcpyDeviceToHost);
	hipFree(d_i);
}
