#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_call.h>
#include <stdio.h>
__global__ void mandelbrot_2(unsigned int* a, unsigned int constw, unsigned int consth, float middlex,
                            float middley, float scale, unsigned int max)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int index = row * constw + col;
  if(col >= constw || row >= consth) return;
    float ci = (row * scale - middley);
    float cr = (col * scale - middlex) ;
    float x = 0;
    float y = 0;
    float xq = 0;
    float yq = 0;
   unsigned int iteration = 0;
  while (iteration++ < max && (xq + yq) < 4)
  {
    y = 2 * x * y + ci;
    x = xq - yq + cr;
    xq = x * x;
    yq = y * y;
  }
  a[index] = iteration;
}


extern "C" void cuda_call_mandelbrot(unsigned int* a_h, unsigned int constw, unsigned int consth, float middlex,
                                    float middley, float scale, unsigned int max, unsigned int reset)
{
  static unsigned int *a_d = NULL;
  static size_t size = 0;
  static dim3 block_size(16, 16);
  static dim3 grid_size(constw / block_size.x + (constw - constw / block_size.x), consth / block_size.y + (consth - consth / block_size.y));
  if (!reset)
  {
    if (size == 0)
    {
      size = constw * consth * sizeof(unsigned int);
      hipMalloc((void **) &a_d, size);
    }
   mandelbrot_2 <<< grid_size, block_size, 0 >>> ((unsigned int *)a_d, constw, consth, middlex, middley, scale, max);
   hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
  }
  else
    hipFree(a_d);
}
